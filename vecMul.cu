#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>


__global__ void matrixVectorMultiply(boolean *a, boolean *b, boolean *c) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < N) {
        float sum = 0.0f;
        for (int i = 0; i < N; ++i) {
            sum += a[tid * N + i] * b[i];
        }
        c[tid] = sum;
    }
}

int *vecMUL(V1,V2) {
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;

    V1_a = (float *)malloc(N * N * sizeof(float));
    V2_a = (float *)malloc(N * N * sizeof(float));

    h_a = (float *)malloc(N * N * sizeof(float));
    h_b = (float *)malloc(N * sizeof(float));
    h_c = (float *)malloc(N * sizeof(float));

    for (int i = 0; i < N * N; ++i) {
        h_a[i] = rand() % 10;
    }
    for (int i = 0; i < N; ++i) {
        h_b[i] = rand() % 10;
    }

    hipMalloc((void **)&V1_a, N * N * sizeof(float));
    hipMalloc((void **)&V2_a, N * N * sizeof(float));
    hipMalloc((void **)&d_a, N * N * sizeof(float));
    hipMalloc((void **)&d_b, N * sizeof(float));
    hipMalloc((void **)&d_c, N * sizeof(float));

    hipMemcpy(d_a, h_a, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockDim(256);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x);

    matrixVectorMultiply<<<gridDim, blockDim>>>(d_a, d_b, d_c);

    hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Result vector:" << std::endl;
    for (int i = 0; i < N; ++i) {
        std::cout << h_c[i] << "\t";
    }
    std::cout << std::endl;

    free(V1_a);
    free(V1_a);
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}



__global__ void vectorMultiplication(float *a, float *b, float *c) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < N) {
        c[tid] = a[tid] * b[tid];
    }
}

int vec2vec() {
    float *h_a, *h_b, *h_c; 
    float *d_a, *d_b, *d_c; 

    h_a = (float *)malloc(N * sizeof(float));
    h_b = (float *)malloc(N * sizeof(float));
    h_c = (float *)malloc(N * sizeof(float));

    for (int i = 0; i < N; ++i) {
        h_a[i] = static_cast<float>(rand() % 10);
        h_b[i] = static_cast<float>(rand() % 10);
    }

    hipMalloc((void **)&d_a, N * sizeof(float));
    hipMalloc((void **)&d_b, N * sizeof(float));
    hipMalloc((void **)&d_c, N * sizeof(float));

    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockDim(256);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x);

    vectorMultiplication<<<gridDim, blockDim>>>(d_a, d_b, d_c);

    hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Result vector:" << std::endl;
    for (int i = 0; i < 10; ++i) {
        std::cout << h_c[i] << "\t";
    }
    std::cout << std::endl;


    return 0;
}

